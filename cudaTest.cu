#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int device_count = 0;
    hipError_t error = hipGetDeviceCount(&device_count);
    
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    if (device_count == 0) {
        printf("No CUDA devices found\n");
    } else {
        printf("Number of CUDA devices: %d\n", device_count);
    }
    
    return 0;
}