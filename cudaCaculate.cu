#include "hip/hip_runtime.h"

 #include<time.h>
#include <stdio.h>
 
__global__ void add(const int a, const int b, int *c)
{
        *c = a + b;
}
 
int main()
{
        clock_t start_t,finish_t;
        double total_t = 0;
        start_t = clock();
        int c;
        int *dev_c; // 定义在设备端的接收数据的指针
        hipError_t cudaStatus;
        //为输入参数和输出参数分配内存
        cudaStatus = hipMalloc((void**)&dev_c, sizeof(int));
        if (cudaStatus != hipSuccess) {
                printf("hipMalloc is failed!\n");
        }
        add<<<1, 1 >>>(2, 7, dev_c);
        cudaStatus = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
                printf(" hipMemcpyDeviceToHost is failed!\n");
        }
        hipFree(dev_c);
        printf("2+7=%d\n", c);
        finish_t = clock();

        total_t = (double)(finish_t - start_t) / CLOCKS_PER_SEC;//将时间转换为秒
        printf("CPU 占用的总时间：%f\n", total_t);
        return 0;
}